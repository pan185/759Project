#include "hip/hip_runtime.h"
#include "solve_jacobi.cuh"
//#include <stdio.h>
__global__ void solve1(double* dx, double* dA, double* db, double* dnextX, int size) {
    //printf("kernel launched!\n");
    double sum = 0;
    for (int j = 0; j < size; j++) {
        if (threadIdx.x != j) {
            sum += dA[threadIdx.x*size+j] * dx[j];
        }
    }
    dnextX[threadIdx.x] = (db[threadIdx.x] - sum) / dA[threadIdx.x*size+threadIdx.x];
}

__global__ void solve2(double* dx, double* dA, double* db, double* dnextX, int size) {
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    //int A_index = tidx;
    
    double sum = 0;
    for (int j = 0; j < size; j++) {
        if (tidx != j) sum += dA[tidx*size + j] * dx[j];
    }
    dnextX[tidx] = (db[tidx] - sum) / dA[tidx*size + tidx];
}

__global__ void solve3(double* dx, double* dA, double* db, double* dnextX, int size) {
    extern __shared__ double shared_dx[];

    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    //write to shared memory
    for(int i=threadIdx.x; i < size; i = i+blockDim.x)
	{
		shared_dx[i] = dx[i];
	}
    __syncthreads();

    double sum = 0;
    for (int j = 0; j < size; j++) {
        if (tidx != j) sum += dA[tidx*size + j] * shared_dx[j];
    }
    dnextX[tidx] = (db[tidx] - sum) / dA[tidx*size + tidx];
}

// loop unrolling 2
__global__ void solve4(double* dx, double* dA, double* db, double* dnextX, int size) {
    extern __shared__ double shared_dx[];

    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    //write to shared memory
    for(int i=threadIdx.x; i < size; i = i+blockDim.x)
	{
		shared_dx[i] = dx[i];
	}
    __syncthreads();

    double sum = 0;
    //unrolling by a factor of 2
    for (int j = 0; j < size; j = j+2) {
        if (tidx != j) sum += dA[tidx*size + j] * shared_dx[j];
        if (tidx != j+1) sum += dA[tidx*size + j+1] * shared_dx[j+1];
    }
    dnextX[tidx] = (db[tidx] - sum) / dA[tidx*size + tidx];
}

// loop unrolling 4
__global__ void solve5(double* dx, double* dA, double* db, double* dnextX, int size) {
    extern __shared__ double shared_dx[];

    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    //write to shared memory
    for(int i=threadIdx.x; i < size; i = i+blockDim.x)
	{
		shared_dx[i] = dx[i];
	}
    __syncthreads();

    double sum = 0;
    //unrolling by a factor of 4
    for (int j = 0; j < size; j = j+4) {
        if (tidx != j) sum += dA[tidx*size + j] * shared_dx[j];
        if (tidx != j+1) sum += dA[tidx*size + j+1] * shared_dx[j+1];
        if (tidx != j+2) sum += dA[tidx*size + j+2] * shared_dx[j+2];
        if (tidx != j+3) sum += dA[tidx*size + j+3] * shared_dx[j+3];
    }
    dnextX[tidx] = (db[tidx] - sum) / dA[tidx*size + tidx];
}