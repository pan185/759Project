#include "hip/hip_runtime.h"
#include "solve_jacobi.cuh"
//#include <stdio.h>
__global__ void solve1(double* dx, double* dA, double* db, double* dnextX, int size) {
    //printf("kernel launched!\n");
    double sum = 0;
    for (int j = 0; j < size; j++) {
        if (threadIdx.x != j) {
            sum += dA[threadIdx.x*size+j] * dx[j];
        }
    }
    dnextX[threadIdx.x] = (db[threadIdx.x] - sum) / dA[threadIdx.x*size+threadIdx.x];
}