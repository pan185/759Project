#include "hip/hip_runtime.h"
#include "solve_jacobi.cuh"
//#include <stdio.h>
__global__ void solve1(double* dx, double* dA, double* db, double* dnextX, int size) {
    //printf("kernel launched!\n");
    double sum = 0;
    for (int j = 0; j < size; j++) {
        if (threadIdx.x != j) {
            sum += dA[threadIdx.x*size+j] * dx[j];
        }
    }
    dnextX[threadIdx.x] = (db[threadIdx.x] - sum) / dA[threadIdx.x*size+threadIdx.x];
}

__global__ void solve2(double* dx, double* dA, double* db, double* dnextX, int size) {
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    //int A_index = tidx;
    
    double sum = 0;
    for (int j = 0; j < size; j++) {
        if (tidx != j) sum += dA[tidx*size + j] * dx[j];
    }
    dnextX[tidx] = (db[tidx] - sum) / dA[tidx*size + tidx];
}

__global__ void solve3(double* dx, double* dA, double* db, double* dnextX, int size) {
    extern __shared__ double shared_dx[];

    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    //write to shared memory
    for(int i=threadIdx.x; i < size; i = i+blockDim.x)
	{
		shared_dx[i] = dx[i];
	}
    __syncthreads();

    double sum = 0;
    for (int j = 0; j < size; j++) {
        if (tidx != j) sum += dA[tidx*size + j] * shared_dx[j];
    }
    dnextX[tidx] = (db[tidx] - sum) / dA[tidx*size + tidx];
}