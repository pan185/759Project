#include "hip/hip_runtime.h"
#include "solve_jacobi.cuh"
//#include <stdio.h>
__global__ void solve1(float* dx, float* dA, float* db, float* dnextX, int size) {
    //printf("kernel launched!\n");
    float sum = 0;
    for (int j = 0; j < size; j++) {
        if (threadIdx.x != j) {
            sum += dA[threadIdx.x*size+j] * dx[j];
        }
    }
    dnextX[threadIdx.x] = (db[threadIdx.x] - sum) / dA[threadIdx.x*size+threadIdx.x];
}

__global__ void solve2(float* dx, float* dA, float* db, float* dnextX, int size) {
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    //int A_index = tidx;
    
    float sum = 0;
    for (int j = 0; j < size; j++) {
        if (tidx != j) sum += dA[tidx*size + j] * dx[j];
    }
    dnextX[tidx] = (db[tidx] - sum) / dA[tidx*size + tidx];
}

__global__ void solve3(float* dx, float* dA, float* db, float* dnextX, int size) {
    extern __shared__ float shared_dx[];

    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    //write to shared memory
    for(int i=threadIdx.x; i < size; i = i+blockDim.x)
	{
		shared_dx[i] = dx[i];
	}
    __syncthreads();

    float sum = 0;
    for (int j = 0; j < size; j++) {
        if (tidx != j) sum += dA[tidx*size + j] * shared_dx[j];
    }
    dnextX[tidx] = (db[tidx] - sum) / dA[tidx*size + tidx];
}

// loop unrolling 2
__global__ void solve4(float* dx, float* dA, float* db, float* dnextX, int size) {
    extern __shared__ float shared_dx[];

    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    //write to shared memory
    for(int i=threadIdx.x; i < size; i = i+blockDim.x)
	{
		shared_dx[i] = dx[i];
	}
    __syncthreads();

    float sum = 0;
    for (int j = 0; j < size; j = j+2) {
        if (tidx != j) sum += dA[tidx*size + j] * shared_dx[j];
        if (tidx != j+1) sum += dA[tidx*size + j+1] * shared_dx[j+1];
    }
    dnextX[tidx] = (db[tidx] - sum) / dA[tidx*size + tidx];
}

__global__ void solve5(float* dx, float* dA, float* db, float* dnextX, int size) {
    extern __shared__ float shared_dx[];

    
}