#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <algorithm>
#include <cstdlib>
#include <cstdio>
#include <string>
#include <cstring>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <stdio.h>

// The std::chrono namespace provides timer functions in C++
#include <chrono>

// std::ratio provides easy conversions between metric units
#include <ratio>

// // Provide some namespace shortcuts
// using std::cout;
// using std::chrono::high_resolution_clock;
// using std::chrono::duration;

//using namespace std;

struct f_mult
{
  template <typename Tuple>
  __host__ __device__
  float operator()(Tuple v)
  {
    return thrust::get<0>(v) * thrust::get<1>(v);
  }
};

struct f_nextx
{
  template <typename Tuple>
  __host__ __device__
  float operator()(Tuple v)
  {
    return ((thrust::get<0>(v) - thrust::get<1>(v)) / thrust::get<2>(v)) + thrust::get<3>(v);
    //nextX[i] = (b[i] - sum) / A[i][i];
  }
};

struct divF: thrust::unary_function<int, int>
{
  int n;

  divF(int n_) : n(n_) {}

  __host__ __device__
  int operator()(int idx)
  {
    return idx / n;
  }
};

struct modF: thrust::unary_function<int, int>
{
  int n;

  modF(int n_) : n(n_) {}

  __host__ __device__
  int operator()(int idx)
  {
    return idx % n;
  }
};

// struct diag_index : public thrust::unary_function<int,int>
// {
//   diag_index(int rows) : rows(rows){}

//   __host__ __device__
//   int operator()(const int index) const
//   {
//       return (index*rows + (index%rows));
//   }

//   const int rows;
// };

struct dmF: thrust::unary_function<int, int>
{
  int n;

  dmF(int n_) : n(n_) {}

  __host__ __device__
  int operator()(int i)
  {
    return i*n+i;
  }
};

typedef thrust::counting_iterator<int> countIt;
typedef thrust::transform_iterator<divF, countIt> columnIt;
typedef thrust::transform_iterator<modF, countIt> rowIt;
typedef thrust::transform_iterator<dmF, countIt> diagIt;


void solve(thrust::device_vector<float>& dx, thrust::device_vector<float>& dA, thrust::device_vector<float>& db,
    thrust::device_vector<float>& dnextX, int size, thrust::device_vector<float>& temp, thrust::device_vector<int>&outkey,
    thrust::device_vector<float>&sum)
{
    // std::cout <<"dA= ";
    // for (int i = 0; i<size*size; i++) {
    //     //printf("%f ", v[i]);
    //     std::cout << dA[i]<<" ";
    // }
    // //printf("\n");
    // std::cout << "\n";

    columnIt cv_begin = thrust::make_transform_iterator(thrust::make_counting_iterator(0), divF(size));
    columnIt cv_end   = cv_begin + (size*size);

    rowIt rv_begin = thrust::make_transform_iterator(thrust::make_counting_iterator(0), modF(size));
    rowIt rv_end   = rv_begin + (size*size);

    diagIt dg_begin = thrust::make_transform_iterator(thrust::make_counting_iterator(0), dmF(size));
    diagIt dg_end   = dg_begin + (size);

    // diagIt dg_begin = thrust::make_transform_iterator(thrust::make_counting_iterator(0),diag_index(size));
    // diagIt dg_end   = dg_begin + (size*size);

    //thrust::device_vector<float> temp(size*size);
    thrust::transform(make_zip_iterator(
                        make_tuple(
                            dA.begin(),
                            thrust::make_permutation_iterator(dx.begin(),rv_begin) ) ),
                        make_zip_iterator(
                        make_tuple(
                            dA.end(),
                            thrust::make_permutation_iterator(dx.end(),rv_end) ) ),
                        temp.begin(),
                        f_mult());

    // thrust::device_vector<int> outkey(size);
    // thrust::device_vector<float> sum(size);
    thrust::reduce_by_key(cv_begin, cv_end, temp.begin(), outkey.begin(), sum.begin());
    //   thrust::transform(v.begin(), v.end(), sum.begin(), v.begin(), thrust::plus<float>());

    // std::cout <<"sum= ";
    // for (int i = 0; i<size; i++) {
    //     //printf("%f ", v[i]);
    //     std::cout << sum[i]<<" ";
    // }
    // //printf("\n");
    // std::cout << "\n";

    // thrust::transform(
    //     make_zip_iterator(
    //     make_tuple(
    //         // dA.begin(),
    //         thrust::make_permutation_iterator(db.begin(),rv_begin),
    //         thrust::make_permutation_iterator(sum.begin(),rv_begin),
    //         thrust::make_permutation_iterator(dA.begin(),dg_begin),
    //         thrust::make_permutation_iterator(dx.begin(),rv_begin) 
    //     ) 
    //     ),
    //     make_zip_iterator(
    //     make_tuple(
    //         thrust::make_permutation_iterator(db.end(),rv_end),
    //         thrust::make_permutation_iterator(sum.end(),rv_end),
    //         thrust::make_permutation_iterator(dA.end(),dg_end),
    //         thrust::make_permutation_iterator(dx.end(),rv_end)
    //     ) 
    //     ),
    //     dnextX.begin(),
    //     f_nextx());
    thrust::transform(
      make_zip_iterator(
      make_tuple(
          // dA.begin(),
          db.begin(),
          sum.begin(),
          thrust::make_permutation_iterator(dA.begin(),dg_begin),
          dx.begin()
      ) 
      ),
      make_zip_iterator(
      make_tuple(
          db.end(),
          sum.end(),
          thrust::make_permutation_iterator(dA.end(),dg_end),
          dx.end()
      ) 
      ),
      dnextX.begin(),
      f_nextx());
    //nextX[i] = ((b[i] - sum) / A[i][i]) + x[i];

    // std::cout <<"nextX= ";
    // for (int i = 0; i<size; i++) {
    //     //printf("%f ", v[i]);
    //     std::cout << dnextX[i]<<" ";
    // }
    // //printf("\n");
    // std::cout << "\n";

}

int main(int argc, char ** argv) {
    int maxIterations = 100;
    int size = std::stoi(argv[1], 0, 10);
    std::cout << "size="<<size<<"\n";
    thrust::host_vector<float> A_flat(size*size);
    thrust::host_vector<float> hb(size);
    thrust::host_vector<float> hx(size);
    thrust::host_vector<float> hnextX(size);

    int n = size;
    std::string rfile = argv[2];
    std::ifstream fin(rfile);
    for (int i = 0; i < n; i++) {
      for (int j = 0; j < n; j++) {
        fin >> A_flat[i*n+j];
        //cout << A[i][j] << " ";
      }
        //cout << endl;
    }
      for (int i = 0; i < n; i++) {
        fin >> hb[i];
      }
      fin.close();
      std::cout << "Read benchmark file "<<rfile<<std::endl;

//   //float * A_flat = new float [size*size];
//   for (int i = 0; i< size*size; i++) {
//       A_flat[i] = i;
//   }
  int size2=size*size;
  thrust::device_vector<float> dA(size2);
  thrust::device_vector<float> dx(size);
  thrust::device_vector<float> db(size);
  thrust::device_vector<float> dnextX(size);
  
  thrust::device_vector<float> temp(size*size);
  thrust::device_vector<int> outkey(size);
  thrust::device_vector<float> sum(size);

  //thrust::fill(dA.begin(), dA.end(), A_flat);
  //thrust::copy(dA.begin(), dA.end(), A_flat);
  dA = A_flat;
  db = hb;
//   thrust::fill(db.begin(), db.end(), 3);
  thrust::fill(dx.begin(), dx.end(), 0);
  thrust::fill(dnextX.begin(), dnextX.end(), 0);

  hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

  int count = 1;
	for (; (count < maxIterations) ; count++)
	{
    if (count % 2) {
			// odd
      solve(dnextX, dA, db, dx, size, temp, outkey, sum);
    }
    else {
      // even
      solve(dx, dA, db, dnextX, size, temp, outkey, sum);
    }
  }

  hipEventRecord(stop);
    hipEventSynchronize(stop);
// Get the elapsed time in milliseconds
float ms = 0;
hipEventElapsedTime(&ms, start, stop);
  std::cout << std::endl << "Iterations:" << count << std::endl;
  printf("%f\n", ms);
  
  hx = dx;
  hnextX = dnextX;

  std::string wfile = argv[3];
  std::ofstream fout(wfile);
    for (int i = 0; i < n; i++)
	{
		fout << std::fixed<<hx[i] << " ";
		//cout << x[i] << "   ";
	}
	fout << std::endl;
	fout.close();

    float * c = new float [size];
	float maxError = 0;
	float total_err = 0;

   for(int i = 0; i < size; i++) {
      c[i] = 0;
      for(int j = 0; j < size; j++)
      {
         c[i] += A_flat[i*size+j] * hx[j];
      }
	  maxError = fmax(maxError, fabs(c[i] - hb[i]));
	  total_err += fabs(c[i] - hb[i]);
   }
   total_err = total_err / size;
   std::cout << "\n==== max error: "<<maxError<<"\n";
	std::cout << "==== avg error: "<<total_err<<"\n";
   delete[] c;

  return 0;
}
