#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include"JacobiGPU.h"
#include  <fstream>
#include<iostream>
#include <stdio.h>
#include <math.h>
#include <cstring>
#include <random>
// The std::chrono namespace provides timer functions in C++
#include <chrono>

// std::ratio provides easy conversions between metric units
#include <ratio>

// Provide some namespace shortcuts
using std::cout;
using std::chrono::high_resolution_clock;
using std::chrono::duration;

using namespace std;


void JacobiGPU::freeAllMemory() {
	freeMemory();
}

void JacobiGPU::solve(double eps) {
	solve_device(eps);
}

void JacobiGPU::solve_host(double eps) {

	high_resolution_clock::time_point start;
    high_resolution_clock::time_point end;
    duration<double, std::milli> duration_sec;
    
	double residual = 0.0;  //	
	double sum = 0.0;
	double dis = 0.0;
	double diff = 1.0;  
	int multicity = int(0.1 / eps);
	//timer.start();
	// Get the starting timestamp
    start = high_resolution_clock::now();

	int count = 1;
	for (; (count < maxIterations) && (diff > eps); count++)
	{
		diff = 0.0;

		for (int i = 0; i < size; i++)
		{
			for (int j = 0; j < size; j++)
			{
				if (i != j)
				{
					sum += A[i][j] * x[j];
				}
			}
			nextX[i] = (b[i] - sum) / A[i][i];

			// if (isnan(nextX[i])) {
			// 	cout << "Not converge"<<endl;
			// 	freeAllMemory();
			// 	exit(EXIT_FAILURE);
			// }

			sum = 0.0;
		}
		residual = 0.0;
		
		for (int m = 0; m < size; m++)
		{
			dis = fabs(nextX[m] - x[m]);
			if (dis > residual)
				residual = dis;
		}
		diff = residual;
		if (diff < eps*multicity) {
			//cout << "======time stop:" << timer.stop() << " ";
			multicity = int(multicity / 10);
		}
		memcpy(x, nextX, size * sizeof(double));
	}
	// Get the ending timestamp
	end = high_resolution_clock::now();
	cout << endl << "Iterations:" << count << endl;
	
    
    // Convert the calculated duration to a double using the standard library
    duration_sec = std::chrono::duration_cast<duration<double, std::milli>>(end - start);
	cout << duration_sec.count() << "\n";

}

void JacobiGPU::solve_device(double eps) {
    
	// double residual = 0.0; 
	// //double sum = 0.0;
	// double dis = 0.0;
	// double diff = 1.0;  
	// int multicity = int(0.1 / eps);
	int numTiles = (size + threads_per_tile - 1) / threads_per_tile;

	cout << "Using GPU kernel "<<kernel_option<<"\n";
	if (kernel_option == 2 || kernel_option == 3) {
		cout << "threads per tile: "<<threads_per_tile<<"\n";
		cout << "number of tiles: "<< numTiles <<"\n";
	}

	int numBlocks = 1;
	int threads_per_block = size;

	// device array allocation
    double *dA;
    hipMalloc((void **)&dA, sizeof(double) * size*size);
    double *db;
    hipMalloc((void **)&db, sizeof(double) * size);
	double *dx;
    hipMalloc((void **)&dx, sizeof(double) * size);
	double *dnextX;
    hipMalloc((void **)&dnextX, sizeof(double) * size);

    hipMemcpy(dA, A_flat, (size*size)*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(db, b, (size)*sizeof(double), hipMemcpyHostToDevice);
	//hipMemcpy(dx, x, (size)*sizeof(double), hipMemcpyHostToDevice);
	//hipMemcpy(dnextX, nextX, (size)*sizeof(double), hipMemcpyHostToDevice);
	hipMemset(dx, 0, size*sizeof(double));
	hipMemset(dnextX, 0, size*sizeof(double));

	hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

	int count = 0;
	//&& (diff > eps)
	for (; (count < maxIterations) ; count++)
	{
		if (count % 2) {
			// odd
			switch(kernel_option) {
				case 1:
				solve1<<<numBlocks, threads_per_block>>>(dnextX, dA, db, dx, size);
				break;

				case 2:
				solve2<<< numTiles, threads_per_tile >>>(dnextX, dA, db, dx, size);
				break;

				case 3:
				solve3<<< numTiles, threads_per_tile, size*sizeof(double) >>>(dnextX, dA, db, dx, size);
				break;

				default:
				solve1<<<numBlocks, threads_per_block>>>(dnextX, dA, db, dx, size);

			}
			
		}
		else {
			// even
			switch(kernel_option) {
				case 1:
				solve1<<<numBlocks, threads_per_block>>>(dx, dA, db, dnextX, size);
				break;

				case 2:
				solve2<<< numTiles, threads_per_tile >>>(dx, dA, db, dnextX, size);
				break;

				case 3:
				solve3<<< numTiles, threads_per_tile, size*sizeof(double) >>>(dx, dA, db, dnextX, size);
				break;
				
				default:
				solve1<<<numBlocks, threads_per_block>>>(dx, dA, db, dnextX, size);

			}
		}

	}
	hipEventRecord(stop);
    hipEventSynchronize(stop);

	hipDeviceSynchronize();
	hipMemcpy(x, dx, sizeof(double) * size, hipMemcpyDeviceToHost);
	hipMemcpy(nextX, dnextX, sizeof(double) * size, hipMemcpyDeviceToHost);
	
	
    // Get the elapsed time in milliseconds
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

	cout << endl << "Iterations:" << count << endl;
	//printf("Iterations:%d\n",count);
    printf("%f\n", ms);

	hipFree(dA); 
	hipFree(db); 
	hipFree(dx); 
	hipFree(dnextX);
	hipEventDestroy(start);
    hipEventDestroy(stop);

}

void JacobiGPU::input(string wfile, bool generate) {
	if (generate) {
		int n = this->size;
		ofstream fout(wfile);

		std::random_device rd;
		std::mt19937 gen(rd());
		std::uniform_real_distribution<> dis(-10, 10);

		for (int i = 0; i < n; i++) {
			for (int j = 0; j < n; j++) {
				if (i == j) {
					this->A_flat[i*n+j] = 10000 * dis(gen);
				}
				else {
					this->A_flat[i*n+j] = dis(gen);
				}
				
				//cout << A[i][j] << " ";
				fout << this->A_flat[i*n+j] << " ";
			}
			//cout << endl;
			fout<< endl;
		}
		
		for (int i = 0; i < n; i++) {
			b[i] = dis(gen)*100;
			//cout << b[i]<<endl;
			fout << b[i]<<" ";
		}
		fout << endl;
		fout.close();
		cout << "Generated random inputs, written to "<<wfile<<endl;
	}
	else {
		//BasicSolver::input(wfile);
		int n = this->size;
		ifstream  fin(wfile);
		for (int i = 0; i < n; i++) {
			for (int j = 0; j < n; j++) {
				fin >> A_flat[i*n+j];
				//cout << A[i][j] << " ";
			}
			//cout << endl;
		}
		for (int i = 0; i < n; i++) {
			fin >> b[i];
		}
		fin.close();
		cout << "Read benchmark file "<<wfile<<endl;
	}
}

void JacobiGPU::mycomputeError() {
	double * c = new double [size];
	double maxError = 0;
	double total_err = 0;

   for(int i = 0; i < size; i++) {
      c[i] = 0;
      for(int j = 0; j < size; j++)
      {
         c[i] += A_flat[i*size+j] * x[j];
      }
	  maxError = fmax(maxError, fabs(c[i] - b[i]));
	  total_err += fabs(c[i] - b[i]);
   }
   total_err = total_err / size;
   cout << "\n==== max error: "<<maxError<<"\n";
	cout << "==== avg error: "<<total_err<<"\n";
   delete[] c;

}

int main(int argc, char ** argv) {
	int dimension = stoi(argv[1], 0, 10);
	bool generate_random = stoi(argv[3], 0, 10);
	int kernel = stoi(argv[6], 0, 10);
	int tpt = stoi(argv[7], 0, 10);
	
	//cout << dimension;
	JacobiGPU * jacobi = new JacobiGPU(dimension);

	jacobi->input(argv[2], generate_random);
	double eps = stod(argv[4]);
	//jacobi->solve_host(eps);
	jacobi->kernel_option = kernel;
	jacobi->threads_per_tile = tpt;

	jacobi->solve(eps);
	jacobi->output(argv[5]);
	jacobi->mycomputeError();
	jacobi->freeAllMemory();
	
}